
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

static void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define HANDLE_NULL(a)                                                         \
  {                                                                            \
    if (a == NULL) {                                                           \
      printf("Host memory failed in %s at line %d\n", __FILE__, __LINE__);     \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

__global__ void saxpy(int *x, int *y, int alpha, size_t N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    y[i] = alpha * y[i] + x[i];
  }
}

void initialize_list(int *x, int N) {
  for (int i = 0; i < N; i++) {
    x[i] = rand();
  }
}

int main(void) {
  int N = std::pow(10, 7);
  int alpha = 2;

  int *x, *y;
  x = (int *)(malloc(N * sizeof(int)));
  y = (int *)(malloc(N * sizeof(int)));

  srand(time(NULL));
  initialize_list(x, N);
  initialize_list(y, N);

  int *d_x, *d_y;
  HANDLE_ERROR( hipMalloc((void **)&d_x, N * sizeof(int)) );
  hipMalloc((void **)&d_y, N * sizeof(int));

  hipMemcpy(d_x, x, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(int), hipMemcpyHostToDevice);

  saxpy<<<(N + 255) / 256, 256>>>(d_x, d_y, alpha, N);

  int *c;
  c = (int *)(malloc(N * sizeof(int)));

  hipMemcpy(c, d_y, N * sizeof(int), hipMemcpyDeviceToHost);

  printf("[");
  for (int i = 0; i < N; i++) {

    if (i < 10) {
      printf("%d ", c[i]);
    }

    if (y[i] * alpha + x[i] != c[i]) {
      printf("YOU SCREWED UP!");
    }
  }
  printf(" ... ]");

  hipFree(d_x);
  return 0;
}
